#include "hip/hip_runtime.h"
/*
	Validate ed25519 implementation against the official test vectors from 
	http://ed25519.cr.yp.to/software.html
*/

#include <stdio.h>
#include <string.h>
#include "ed25519.h"

#include "test-ticks.h"

#include "ed25519.cu"

static void
edassert(int check, int round, const char *failreason) {
	if (check)
		return;
	printf("round %d, %s\n", round, failreason);
	exit(1);
}

static void
edassert_die(const unsigned char *a, const unsigned char *b, size_t len, int round, const char *failreason) {
	size_t i;
	if (round > 0)
		printf("round %d, %s\n", round, failreason);
	else
		printf("%s\n", failreason);
	printf("want: "); for (i = 0; i < len; i++) printf("%02x,", a[i]); printf("\n");
	printf("got : "); for (i = 0; i < len; i++) printf("%02x,", b[i]); printf("\n");
	printf("diff: "); for (i = 0; i < len; i++) if (a[i] ^ b[i]) printf("%02x,", a[i] ^ b[i]); else printf("  ,"); printf("\n\n");
	exit(1);
}

static void
edassert_equal(const unsigned char *a, const unsigned char *b, size_t len, const char *failreason) {
	if (memcmp(a, b, len) == 0)
		return;
	edassert_die(a, b, len, -1, failreason);
}

static void
edassert_equal_round(const unsigned char *a, const unsigned char *b, size_t len, int round, const char *failreason) {
	if (memcmp(a, b, len) == 0)
		return;
	edassert_die(a, b, len, round, failreason);
}


/* test data */
typedef struct test_data_t {
	unsigned char sk[32], pk[32], sig[64];
	char m[1024];
} test_data;


test_data dataset[] = {
#include "regression.h"
};

/* test data with much shorter messages */
#define MIN_MESSAGE_LEN (135)
#define MAX_MESSAGE_LEN (144)
typedef struct short_test_data_t {
	unsigned char sk[32], pk[32], sig[64];
	char m[((MAX_MESSAGE_LEN + 16 - 1)/16) * 16];
} short_test_data;

#define MESSAGES_UNDER_TEST (16384)
int interesting_dataset_msglen[MESSAGES_UNDER_TEST];
short_test_data interesting_dataset[MESSAGES_UNDER_TEST];

/* result of the curve25519 scalarmult ((|255| * basepoint) * basepoint)... 1024 times */
const curved25519_key curved25519_expected = {
	0xac,0xce,0x24,0xb1,0xd4,0xa2,0x36,0x21,
	0x15,0xe2,0x3e,0x84,0x3c,0x23,0x2b,0x5f,
	0x95,0x6c,0xc0,0x7b,0x95,0x82,0xd7,0x93,
	0xd5,0x19,0xb6,0xf1,0xfb,0x96,0xd6,0x04
};


/* from ed25519-donna-batchverify.h */
extern unsigned char batch_point_buffer[3][32];

/* y coordinate of the final point from 'amd64-51-30k' with the same random generator */
static const unsigned char batch_verify_y[32] = {
	0x51,0xe7,0x68,0xe0,0xf7,0xa1,0x88,0x45,
	0xde,0xa1,0xcb,0xd9,0x37,0xd4,0x78,0x53,
	0x1b,0x95,0xdb,0xbe,0x66,0x59,0x29,0x3b,
	0x94,0x51,0x2f,0xbc,0x0d,0x66,0xba,0x3f
};

/*
static const unsigned char batch_verify_y[32] = {
	0x5c,0x63,0x96,0x26,0xca,0xfe,0xfd,0xc4,
	0x2d,0x11,0xa8,0xe4,0xc4,0x46,0x42,0x97,
	0x97,0x92,0xbe,0xe0,0x3c,0xef,0x96,0x01,
	0x50,0xa1,0xcc,0x8f,0x50,0x85,0x76,0x7d
};

Introducing the 128 bit r scalars to the heap _before_ the largest scalar
fits in to 128 bits alters the heap shape and produces a different,
yet still neutral/valid y/z value.

This was the value of introducing the r scalars when the largest scalar fit
in to 135-256 bits. You can produce it with amd64-64-24k / amd64-51-32k
with the random sequence used in the first pass by changing

    unsigned long long hlen=((npoints+1)/2)|1;

to

    unsigned long long hlen=npoints;

in ge25519_multi_scalarmult.c

ed25519-donna-batchverify.h has been modified to match the 
default amd64-64-24k / amd64-51-32k behaviour
*/



/* batch test */
#define test_batch_count 64
#define test_batch_rounds 96

typedef enum batch_test_t {
	batch_no_errors = 0,
	batch_wrong_message = 1,
	batch_wrong_pk = 2,
	batch_wrong_sig = 3
} batch_test;

void EDKERNEL
ed25519_sign_open_kernel(short_test_data* data, int* msg_size, unsigned char* result, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int r;
	if (i>= N) {
		return ;
	}
#if 0
	if (i < 2) {
# define PP(bb) printf("%d: " #bb " xyz: %d %d %d\n", i, bb.x, bb.y, bb.z)
		PP(threadIdx);
		PP(blockIdx);
		PP(blockDim);
	}
#endif
	//printf("GPU at %d\n", i);
	r = ed25519_sign_open((unsigned char*)data[i].m, msg_size[i], data[i].pk, data[i].sig);
	//printf("GPU result at %d is %d\n", i, r);
	result[i] = r;
}

static void
test_main_CUDA(void) {
	int i, j, N = 5, res;
	ed25519_public_key pk;
	ed25519_signature sig;
	unsigned char forge[1024] = {'x'};
	unsigned char results[MESSAGES_UNDER_TEST], *gpu_results;
	int* lengths;
	short_test_data* gpu_test_data;
	curved25519_key csk[2] = {{255}};
	uint64_t ticks, pkticks = maxticks, signticks = maxticks, openticks = maxticks, curvedticks = maxticks;

	for (i=0, j = MIN_MESSAGE_LEN;i<MESSAGES_UNDER_TEST; i++) {
		memcpy(&interesting_dataset[i], &dataset[j], sizeof(interesting_dataset[i]));
		interesting_dataset_msglen[i] = j;
		j ++;
		if (j > MAX_MESSAGE_LEN) {
			j = MIN_MESSAGE_LEN;
		}
	}
	ticks = get_ticks();
	for (j=0;j<N;j++) {
		for (i = 0; i < MESSAGES_UNDER_TEST; i++) {
			results[i] = ed25519_sign_open((unsigned char *)interesting_dataset[i].m, interesting_dataset_msglen[i], interesting_dataset[i].pk, interesting_dataset[i].sig);
		}
	}
	ticks = get_ticks() - ticks;
	printf("%.0f ticks to verify %d * %d signatures on CPU, data set size %zu, one message time %.0f\n", (double)ticks, MESSAGES_UNDER_TEST, N, sizeof(interesting_dataset), ((double)ticks)/(N * MESSAGES_UNDER_TEST));

	int block_size = 256;
        int num_blocks = (MESSAGES_UNDER_TEST + (block_size) - 1) / (block_size);
	CUCHK(hipMallocManaged(&lengths, sizeof(*lengths) * MESSAGES_UNDER_TEST));
	memcpy(lengths, interesting_dataset_msglen, sizeof(*lengths) * MESSAGES_UNDER_TEST);

	CUCHK(hipMallocManaged(&gpu_test_data, sizeof(interesting_dataset)));
	CUCHK(hipMallocManaged(&gpu_results, sizeof(*gpu_results) * MESSAGES_UNDER_TEST));
	memset(gpu_results, 111, sizeof(*gpu_results) * MESSAGES_UNDER_TEST);
	ticks = get_ticks();
	for (j=0;j<N;j++) {
		//CUCHK(hipMemcpy(gpu_test_data, interesting_dataset, sizeof(interesting_dataset), hipMemcpyHostToDevice));
		memcpy(gpu_test_data, interesting_dataset, sizeof(interesting_dataset));
	        ed25519_sign_open_kernel<<<num_blocks,block_size>>>(gpu_test_data, lengths, gpu_results, MESSAGES_UNDER_TEST);
		CUCHK(hipPeekAtLastError());
		CUCHK(hipDeviceSynchronize());
		for (i = 0;i < MESSAGES_UNDER_TEST;i ++) {
			if (gpu_results[i] != results[i]) {
				printf("difference %d/%d at %d\n", gpu_results[i], results[i], i);
				break;
			}
		}
	}
	ticks = get_ticks() - ticks;
	printf("%.0f ticks to verify %d * %d signatures on GPU, %.0f ticks per message\n", (double)ticks, MESSAGES_UNDER_TEST, N, ((double)ticks)/(MESSAGES_UNDER_TEST * N));

}

int
main(void) {
	test_main_CUDA();
	return 0;
}

